/*
*
* Addition on the Device: add()
*
*/




#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(int *a, int *b, int *c)
{
	*c = *a + *b;
}

//main
int main(void){
	int a, b, c;  //host copies of a,b,c
	int *d_a, *d_b, *d_c; //device copies of a,b,c
	int size = sizeof(int);

	//allocating space for device copies of a,b,c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	//setup input values: initializing our input data
	a = 2;
	b = 7;

	//copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	//launch add() kernel in GPU: pass arrgument like a normal function
	add<<<1,1>>>(d_a, d_b, d_c);

	//copy result back to host
	hipMemcpy (&c, d_c, size, hipMemcpyDeviceToHost);

	//cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;

}

